#include "hip/hip_runtime.h"
// reading a text file
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <math.h>
#include "define.h"

#define MATCH       +1
#define MISMATCH    -1
#define GAP         -1
using namespace std;

void FillMatrix(const string *sequence, int n, int *scoring)
{
  int rows,cols;
  int *c;
  for(int a = 0; a< n; a++){
    for(int b = 0; b < n; b++){
      string X = sequence[a];
      string Y = sequence[b];
      rows = X.length();
      cols = Y.length();
      c = (int*)malloc(((cols + 1)*(rows + 1)) * sizeof(int));

      int score = 0;
        int letak,kiri,atas,miring, n_letak, n_atas, n_kiri,x=0,y=0,i;
        int jum = (cols + 1)*(rows + 1);
      
        for(i = 0; i< jum;i++)
        {   
          if(i > (cols + 1 )  && (i % (cols+1) != 0))
          {
          y = (i/(cols + 1) - 1);
          x = (i-1) % (cols + 1);
      
          letak = i ;
          kiri  = letak - 1;
          atas  = (letak - cols) - 1 ;
          miring = atas - 1;    
                //c[letak] = atas;
                //printf("%d - %d = %.0f \n", X[y]  - 'A', Y[x]  - 'A' , scoringsMatrixHost[X[y] - 'A'][Y[x] - 'A']);
                n_letak = c[miring] + scoringsMatrixHost[X[y] - 'A'][Y[x] - 'A'];
                n_kiri = c[kiri] + GAP;
                n_atas = c[atas] + GAP;
                //printf("MIRING %d - KIRI %d - ATAS %d\n", c[miring], c[kiri], c[atas]);
          
                if (n_letak > n_atas && n_letak > n_kiri && n_letak > 0) {
                  c[letak] = n_letak;
                }
                else if (n_atas > n_kiri && n_atas > 0) {
                  c[letak] = n_atas;
                }
                else if (n_kiri > 0 ){
                  c[letak] = n_kiri;
                }else {
                  c[letak] = 0;
                }
                if(score < c[letak]){
                  score = c[letak];
                }
            }
            else{
                c[i] = 0;
            }
        } 
        scoring[(a*(n)) + b] = score;
        //cout << "Score "<< a <<"-"<<b<<" : " << score << "\n";
        free(c);
        //return score;
    }
  }
}

int main (int argc, char **argv) {
  string line,x,y,line2;
  int i = 0, n;
  int *score;

  vector<string> sequence_id;
  vector<string>::iterator it;

  ifstream myfile ("prot_list.csv");
  if (myfile.is_open())
  {
    while ( getline (myfile,line) )
    { 
        sequence_id.push_back(line);
    }
    
    myfile.close();
  }
  //else cout << "Unable to open file";
  cin >> n;
  if(n == 0)
    n = sequence_id.size();
    int size = sequence_id.size();
    string *sequence =  new string[size];

    score    = (int*)malloc((n *n) * sizeof(int));
    
    for(it = sequence_id.begin(); it < sequence_id.end(); it++){
        ifstream myfile2 ("Fasta/" + *it + ".fasta");
        if (myfile2.is_open())
        {
            x="";
            while ( getline (myfile2,line2) )
            {
            if(line2[0] != '>')  
                    x += line2;
            }
            sequence[i] = x;
            //cout << "X : " <<  x << "\n"; 
            myfile2.close();
        }else{
          sequence_id.erase(it);
        }
        //else cout << *it << " Unable to open file \n"; 
        i++;
    }

    cout << sequence_id.size();
    clock_t start, end;
    start = clock();
    FillMatrix(sequence,n,score);
    end = clock();
  	double extime = (double)(end - start) / CLOCKS_PER_SEC;
    //free(score);free(sequence);

    ofstream savefile (argv[1]);
    if (savefile.is_open())
    {
      for(int i = 0; i< n ; i++){
        for(int j = 0; j < n; j++){
          savefile << sequence_id.at(i) << "," << sequence_id.at(j) << "," << score[(i*(n)) + j]  << "\n";
        }
      }
      savefile.close();
    }
    else cout << "Unable to open file";
    //free(score);free(sequence);
    printf("\nwaktu : %f seconds\n ", extime);
    return 0;

}