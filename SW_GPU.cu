#include "hip/hip_runtime.h"
// reading a text file
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <math.h>
#include "define.h"

#define MATCH       +1
#define MISMATCH    -1
#define GAP         -1
using namespace std;

__global__ void FillMatrix(char **sequence,int *s_length,int n, short int *score,short int **c)
{
  int a = threadIdx.x + blockIdx.x*blockDim.x;
  int b = threadIdx.y + blockIdx.y*blockDim.y;
  int letak,kiri,atas,miring, n_letak, n_atas, n_kiri,x=0,y=0,i;
  const int rows = s_length[a],cols = s_length[b];
    //printf("Masuk %d, %d\n",a,b);
      char *X = sequence[a];
      char *Y = sequence[b];
      
       const int jum = (cols + 1)*(rows + 1);
       
      //printf(" got C pointer: %p\n" , c[(a*(n)) + b]);
      score[(a*(n)) + b] = 0;
      if(b < n && a < n){
        if(b >= a){
          for(i = 0; i < jum;i++)
          {   
            if(i > (cols + 1 )  && (i % (cols+1) != 0))
            {
                y = (i/(cols + 1) - 1);
                x = (i-1) % (cols + 1);
            
                letak = i ;
                kiri  = letak - 1;
                atas  = (letak - cols) - 1 ;
                miring = atas - 1;
                int scoring = scoringsMatrix[X[y] - 'A'][Y[x] - 'A'];    
                n_letak = c[(a*(n)) + b][miring] + scoring;
                n_kiri = c[(a*(n)) + b][kiri] + GAP;
                n_atas = c[(a*(n)) + b][atas] + GAP;
                //c[(a*(n)) + b][miring] = i;
                //c[(a*(n)) + b][letak] = 5;
                if (n_letak > n_atas && n_letak > n_kiri && n_letak > 0) {
                  c[(a*(n)) + b][letak] = n_letak;
                }
                else if (n_atas > n_kiri && n_atas > 0) {
                  c[(a*(n)) + b][letak] = n_atas;
                }
                else if (n_kiri > 0 ){
                  c[(a*(n)) + b][letak] = n_kiri;
                }else {
                  c[(a*(n)) + b][letak] = 0;
                }
                if(score[(a*(n)) + b] < c[(a*(n)) + b][letak]){
                  score[(a*(n)) + b] = c[(a*(n)) + b][letak];
                }
                
            }
            else{
              c[(a*(n)) + b][i] = 0;
            }
          }
        }
      }
}

int main (int argc, char **argv) {
  hipError_t err = hipSuccess;
  string line,y, line2, x;
  int i = 0, n,j;

  int *len;
  short int *score;
  short int *dscore;
  int *dlen;

  vector<string> sequence_id;
  vector<string>::iterator it;

  

  ifstream myfile (argv[2]);
  if (myfile.is_open())
  {
    while ( getline (myfile,line) )
    { 
        sequence_id.push_back(line);
    }
    
    myfile.close();
  }
  int size = sequence_id.size();
  n = atoi(argv[3]);
  cout << "Jumlah data : " << n << "\n";
  
  if(n == 0)
    n = size;
     char **sequence = new  char*[size];
     char **dsequence = new  char*[size];

     score    = (short int*)malloc((n *n) * sizeof(short int));
     len    = (int*)malloc(size * sizeof(int));
     
    for(it = sequence_id.begin(); it < sequence_id.end(); it++){
        ifstream myfile2 ("Fasta/" + *it + ".fasta");
        if (myfile2.is_open())
        {
            x = "";
            while ( getline (myfile2,line2) )
            {
            if(line2[0] != '>')  
                    x += line2;
            }
            char *tem = new char[x.length()];
            strcpy(tem,x.c_str());
            sequence[i] = tem;
            delete []tem;
            len[i] = x.length();
            myfile2.close();
        }else{
          sequence_id.erase(it);
        }
        i++;
    }



    hipMalloc((void**)&dscore, (n *n) * sizeof(short int));
    hipMalloc((void**)&dlen,size * sizeof(int));

    hipMallocManaged(&dsequence, size*sizeof(char *));
  
    // initialize dynamic array array
    for (int i = 0; i < size; i++)
    {
      hipMallocManaged(&(dsequence[i]), len[i]*sizeof(char));
      memcpy(dsequence[i], sequence[i], len[i]);
    }

    err = hipMemcpy(dscore, score, ((n) *  (n)) * sizeof(short int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dlen, len,size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    short int **dc;
    hipMallocManaged(&dc, (n + 1) * (n + 1) * sizeof(short int *));
    for(int i = 0; i< n ; i++){
      for(int j = 0; j < n; j++){
         hipMallocManaged(&(dc[(i*(n)) + j]), (len[i] + 1) * (len[j] + 1) *sizeof(short int));
      }
    }
    
    int t = atoi(argv[4]);
    cout << "Jumlah Thread : " << t << "\n";
    
    //FillMatrix(sequence,n);
    float elapsed=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

     hipEventRecord(start, 0);

    dim3 threadsPerBlock(t,t);
    int block = n/t!=0?n/t:1;
    block += n%t>0?1:0;
    cout << "Jumlah Block : " << block << "\n" << "Start Calculation \n";
    dim3 blocksPerGrid(block,block);
    int threadsPerLunch((t * t) * (block * block));
    //for(j= 0; j < threadsPerBlock; j++){
    //hipDeviceSetLimit(hipLimitMallocHeapSize, threadsPerLunch * (35000 * 35000) * sizeof(int));
    FillMatrix <<<blocksPerGrid, threadsPerBlock>>>(dsequence,dlen,n,dscore,dc);
    //hipDeviceSynchronize();
    //}
    hipEventRecord(stop, 0);
    hipEventSynchronize (stop);

    hipEventElapsedTime(&elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);


    err = hipMemcpy(score, dscore, (n*n) * sizeof(short int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy score from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(dsequence); hipFree(dscore); hipFree(dlen);
    

    ofstream savefile (argv[1]);
    if (savefile.is_open())
    {
      for(int i = 0; i< n ; i++){
        for(int j = 0; j < n; j++){
          if(j >= i)
            savefile << sequence_id.at(i) << "," << sequence_id.at(j) << "," << score[(i*(n)) + j]  << "\n";
          else
            savefile << sequence_id.at(i) << "," << sequence_id.at(j) << "," << score[(j*(n)) + i]  << "\n";
        }
      }
      //savefile << "Waktu Eksekusi " <<  extime; 
      savefile.close();
    }
    else cout << "Unable to open file";
    free(score);free(len);delete[] sequence;
    
    printf("\nwaktu : %f seconds\n ", elapsed/1000);
    return 0;
}